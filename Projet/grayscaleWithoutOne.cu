#include "hip/hip_runtime.h"
﻿#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
#include "hip/hip_runtime.h"
#include ""
#endif

#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>

using namespace std;

__global__ void greyscaleWithoutOneKernel(unsigned char* rgb, unsigned char* g, const size_t cols, const size_t rows)
{
	auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
	auto tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if (tidx < cols && tidy < rows)
	{
		if (50 <= rgb[3 * (tidy * cols + tidx)] && rgb[3 * (tidy * cols + tidx)] <= 200
			&& 20 <= rgb[3 * (tidy * cols + tidx) + 1] && rgb[3 * (tidy * cols + tidx) + 1] <= 180
			&& 10 <= rgb[3 * (tidy * cols + tidx) + 2] && rgb[3 * (tidy * cols + tidx) + 2] <= 160) {

			g[3 * (tidy * cols + tidx)] = (
				307 * rgb[3 * (tidy * cols + tidx)]
				+ 604 * rgb[3 * (tidy * cols + tidx) + 1]
				+ 113 * rgb[3 * (tidy * cols + tidx) + 2]
				) / 1024;
			g[3 * (tidy * cols + tidx) + 1] = (
				307 * rgb[3 * (tidy * cols + tidx)]
				+ 604 * rgb[3 * (tidy * cols + tidx) + 1]
				+ 113 * rgb[3 * (tidy * cols + tidx) + 2]
				) / 1024;
			g[3 * (tidy * cols + tidx) + 2] = (
				307 * rgb[3 * (tidy * cols + tidx)]
				+ 604 * rgb[3 * (tidy * cols + tidx) + 1]
				+ 113 * rgb[3 * (tidy * cols + tidx) + 2]
				) / 1024;
		}

		if (g[3 * (tidy * cols + tidx)] == 0 && g[3 * (tidy * cols + tidx) + 1] == 0 && g[3 * (tidy * cols + tidx) + 2] == 0) {
			g[3 * (tidy * cols + tidx)] = rgb[3 * (tidy * cols + tidx)];
			g[3 * (tidy * cols + tidx) + 1] = rgb[3 * (tidy * cols + tidx) + 1];
			g[3 * (tidy * cols + tidx) + 2] = rgb[3 * (tidy * cols + tidx) + 2];
		}
	}
}


void grayscaleWithoutOne()
{
	cv::Mat m_in = cv::imread("ecureuil.jpg", cv::IMREAD_UNCHANGED);

	auto rgb = m_in.data;
	auto rows = m_in.rows;
	auto cols = m_in.cols;

	std::vector< unsigned char > g(3 * rows * cols);
	cv::Mat m_out(rows, cols, CV_8UC3, g.data());

	unsigned char* rgb_d;
	unsigned char* g_d;

	auto start = std::chrono::system_clock::now();
	hipEvent_t cudaStart, cudaStop;
	hipEventCreate(&cudaStart);
	hipEventCreate(&cudaStop);

	hipEventRecord(cudaStart);

	hipMalloc(&rgb_d, 3 * rows * cols);
	hipMalloc(&g_d, 3 * rows * cols);

	hipMemcpy(rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice);

	dim3 block(32, 32);
	dim3 grid((cols - 1) / block.x + 1, (rows - 1) / block.y + 1); //(4,4)

	cout << "rows : " << rows << endl;
	cout << "cols : " << cols << endl;

	greyscaleWithoutOneKernel << <grid, block >> > (rgb_d, g_d, cols, rows);


	hipMemcpy(g.data(), g_d, 3 * rows * cols, hipMemcpyDeviceToHost);

	hipEventRecord(cudaStop);
	hipEventSynchronize(cudaStop);
	auto stop = std::chrono::system_clock::now();


	auto duration = stop - start;
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, cudaStart, cudaStop);
	std::cout << "Temps kernel: " << elapsedTime << std::endl;
	hipEventDestroy(cudaStart);
	hipEventDestroy(cudaStop);
	auto err = hipGetLastError();

	std::cout << "Erreur: " << err << std::endl;

	std::cout << ms << " ms" << std::endl;

	cv::imwrite("out.jpg", m_out);

	hipFree(rgb_d);
	hipFree(g_d);
}