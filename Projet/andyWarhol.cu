#include "hip/hip_runtime.h"
﻿#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
#include "hip/hip_runtime.h"
#include ""
#endif

#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>
#include "utils.cuh"

using namespace std;

__global__ void duplicateImageWarhol(unsigned char* in, unsigned char* out, std::size_t cols, std::size_t rows, int duplicationNumber = 4)
{
	auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
	auto tidy = blockIdx.y * blockDim.y + threadIdx.y;

	if (tidx < cols && tidy < rows)
	{
		int square = cuSquare(duplicationNumber);

		//For each line
		for (int i = 0; i < square; i++) {

			auto index = 3 * (tidy * cols / square + tidx / square) + 3 * cols * rows / square * i;

			if (out[index] == 0 && out[index + 1] == 0 && out[index + 2] == 0) {
				out[index] = in[3 * (tidy * cols + tidx)];
				out[index + 1] = in[3 * (tidy * cols + tidx) + 1];
				out[index + 2] = in[3 * (tidy * cols + tidx) + 2];
			}
		}
	}
}


__global__ void colorizeImageWarhol(unsigned char* in, unsigned char* out, std::size_t cols, std::size_t rows, int duplicationNumber = 4)
{
	//auto tidx = blockIdx.x * blockDim.x + threadIdx.x;
	//auto tidy = blockIdx.y * blockDim.y + threadIdx.y;

	//if (tidx < cols && tidy < rows)
	//{
	//	int square = cuSquare(duplicationNumber);

	//	//For each line
	//	for (int i = 0; i < square; i++) {

	//		auto index = 3 * (tidy * cols / square / 2 + tidx / square) + 3 * cols * rows / square * i;

	//		if (out[index] == 0 && out[index + 1] == 0 && out[index + 2] == 0) {
	//			out[index] = in[3 * (tidy * cols + tidx)];
	//			out[index + 1] = in[3 * (tidy * cols + tidx) + 1];
	//			out[index + 2] = in[3 * (tidy * cols + tidx) + 2];
	//		}
	//	}
	//}
}


void andyWarhol()
{
	cv::Mat m_in = cv::imread("ecureuil.jpg", cv::IMREAD_UNCHANGED);

	auto rgb = m_in.data;
	auto rows = m_in.rows;
	auto cols = m_in.cols;

	std::vector< unsigned char > duplicated(3 * rows * cols);
	std::vector< unsigned char > out(3 * rows * cols);
	cv::Mat image_duplicated(rows, cols, CV_8UC3, duplicated.data());
	cv::Mat image_out(rows, cols, CV_8UC3, out.data());

	unsigned char* base_d;
	unsigned char* duplicated_d;
	unsigned char* out_d;

#pragma region Event & Timer

	auto start = std::chrono::system_clock::now();
	hipEvent_t cudaStart, cudaStop;
	hipEventCreate(&cudaStart);
	hipEventCreate(&cudaStop);

	hipEventRecord(cudaStart);

#pragma endregion

	hipMalloc(&base_d, 3 * rows * cols);
	hipMalloc(&duplicated_d, 3 * rows * cols);
	hipMalloc(&out_d, 3 * rows * cols);

	hipMemcpy(base_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice);

	dim3 block(16, 64);
	dim3 grid((cols - 1) / block.x + 1, (rows - 1) / block.y + 1); //(4,4)

	cout << "rows : " << rows << endl;
	cout << "cols : " << cols << endl;

	duplicateImageWarhol << <grid, block >> > (base_d, duplicated_d, cols, rows);
	colorizeImageWarhol << <grid, block >> > (duplicated_d, out_d, cols, rows);

	hipMemcpy(duplicated.data(), duplicated_d, 3 * rows * cols, hipMemcpyDeviceToHost);
	hipMemcpy(out.data(), out_d, 3 * rows * cols, hipMemcpyDeviceToHost);

#pragma region Event & Timer


	hipEventRecord(cudaStop);
	hipEventSynchronize(cudaStop);
	auto stop = std::chrono::system_clock::now();

	auto duration = stop - start;
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, cudaStart, cudaStop);
	std::cout << "Temps kernel: " << elapsedTime << std::endl;
	hipEventDestroy(cudaStart);
	hipEventDestroy(cudaStop);
	auto err = hipGetLastError();

	std::cout << "Erreur: " << err << std::endl;

	std::cout << ms << " ms" << std::endl;

#pragma endregion

	cv::imwrite("duplicated.jpg", image_duplicated);
	cv::imwrite("out.jpg", image_out);

	hipFree(base_d);
	hipFree(duplicated_d);
	hipFree(out_d);
}